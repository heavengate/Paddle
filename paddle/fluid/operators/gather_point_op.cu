#include "hip/hip_runtime.h"
/* Copyright (c) 2019 PaddlePaddle Authors. All Rights Reserved.
Licensed under the Apache License, Version 2.0 (the "License");
you may not use this file except in compliance with the License.
You may obtain a copy of the License at
    http://www.apache.org/licenses/LICENSE-2.0
Unless required by applicable law or agreed to in writing, software
distributed under the License is distributed on an "AS IS" BASIS,
WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
See the License for the specific language governing permissions and
limitations under the License. */

#include "paddle/fluid/framework/eigen.h"
#include "paddle/fluid/framework/op_registry.h"

namespace paddle {
namespace operators {

using Tensor = framework::Tensor;

template <typename T>
__global__ void GatherPointKernel(int b, int n, int m,
                                  const T *__restrict__ inp,
                                  const int *__restrict__ idx,
                                  T *__restrict__ out) {
  for (int i = blockIdx.x; i < b; i += gridDim.x) {
    for (int j = blockIdx.y * blockDim.x + threadIdx.x; j < m;
         j += blockDim.x * gridDim.y) {
      int a = idx[i * m + j];
      out[(i * m + j) * 3 + 0] = inp[(i * n + a) * 3 + 0];
      out[(i * m + j) * 3 + 1] = inp[(i * n + a) * 3 + 1];
      out[(i * m + j) * 3 + 2] = inp[(i * n + a) * 3 + 2];
    }
  }
}

template <typename T>
class GatherPointOpCUDAKernel : public framework::OpKernel<T> {
 public:
  void Compute(const framework::ExecutionContext &ctx) const override {
    PADDLE_ENFORCE(platform::is_gpu_place(ctx.GetPlace()),
                   "This kernel only runs on GPU device.");
    auto *points = ctx.Input<Tensor>("X");
    auto *index = ctx.Input<Tensor>("Index");
    auto *output = ctx.Output<Tensor>("Output");
    if (points->numel() == 0) return;
    // allocate memory
    output->mutable_data<T>(ctx.GetPlace());

    int batch_size = points->dims()[0];
    int n_points = points->dims()[1];
    int m_points = index->dims()[1];

    // faltten
    auto in_points = framework::EigenVector<T>::Flatten(*points);
    const T *p_points = &(in_points(0));

    auto in_index = framework::EigenVector<int>::Flatten(*index);
    const int *p_index = &(in_index(0));

    auto out_points = framework::EigenVector<T>::Flatten(*output);
    T *p_out_points = &(out_points(0));

    GatherPointKernel<<<dim3(2, 8, 1), 512>>>(batch_size, n_points, m_points,
                                              p_points, p_index, p_out_points);
  }
};

}  // namespace operators
}  // namespace paddle

namespace ops = paddle::operators;
REGISTER_OP_CUDA_KERNEL(gather_point, ops::GatherPointOpCUDAKernel<float>,
                        ops::GatherPointOpCUDAKernel<double>,
                        ops::GatherPointOpCUDAKernel<int>);
